#include "hip/hip_runtime.h"
#include <string.h>
#include <ctype.h>
#ifndef _WIN32
#include <unistd.h>
#endif

#include "miner.h"

extern "C" char *device_name[8];
extern "C" int device_map[8];
extern "C" long device_sm[8];

extern "C" int cuda_num_devices()
{
    int version;
    hipError_t err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < 5 || (maj == 5 && min < 5))
    {
        applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", 5, 5);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

extern "C" void cuda_devicenames()
{
    hipError_t err;
    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }

    for (int i=0; i < GPU_N; i++)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device_map[i]);

        device_name[i] = strdup(props.name);
		device_sm[i] = (props.major * 100 + props.minor * 10);
    }
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
    int hlen = strlen(haystack);
    int nlen = strlen(needle);
    for (int i=0; i < hlen; ++i)
    {
        if (haystack[i] == ' ') continue;
        int j=0, x = 0;
        while(j < nlen)
        {
            if (haystack[i+x] == ' ') {++x; continue;}
            if (needle[j] == ' ') {++j; continue;}
            if (needle[j] == '#') return ++match == needle[j+1]-'0';
            if (tolower(haystack[i+x]) != tolower(needle[j])) break;
            ++j; ++x;
        }
        if (j == nlen) return true;
    }
    return false;
}

extern "C" int cuda_finddevice(char *name)
{
    int num = cuda_num_devices();
    int match = 0;
    for (int i=0; i < num; ++i)
    {
        hipDeviceProp_t props;
        if (hipGetDeviceProperties(&props, i) == hipSuccess)
            if (substringsearch(props.name, name, match)) return i;
    }
    return -1;
}
