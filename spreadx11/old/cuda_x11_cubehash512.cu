#include "hip/hip_runtime.h"

typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return __byte_perm(x, 0, 0x0123);
}

typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

#define CUBEHASH_ROUNDS 16 /* this is r for CubeHashr/b */
#define CUBEHASH_BLOCKBYTES 32 /* this is b for CubeHashr/b */

typedef unsigned int uint32_t; /* must be exactly 32 bits */

static __device__ __forceinline__ uint32_t ROTL32( uint32_t x, const int n )
{
#if __CUDA_ARCH__ >= 320
    return __funnelshift_l(x, x, n);
#else
    return (x << n) | (x >> (32-n));
#endif
}

#define ROTATEUPWARDS7(a) ROTL32((a), 7)
#define ROTATEUPWARDS11(a) ROTL32((a), 11)
#define SWAP(a,b) { uint32_t u = a; a = b; b = u; }

__constant__ uint32_t c_IV_512[32];
static const uint32_t h_IV_512[32] = {
	0x2AEA2A61, 0x50F494D4, 0x2D538B8B,
	0x4167D83E, 0x3FEE2313, 0xC701CF8C,
	0xCC39968E, 0x50AC5695, 0x4D42C787,
	0xA647A8B3, 0x97CF0BEF, 0x825B4537,
	0xEEF864D2, 0xF22090C4, 0xD0E5CD33,
	0xA23911AE, 0xFCD398D9, 0x148FE485,
	0x1B017BEF, 0xB6444532, 0x6A536159,
	0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
	0xD65C8A2B, 0xA5A70E75, 0xB1C62456,
	0xBC796576, 0x1921C8F7, 0xE7989AF1,
	0x7795D246, 0xD43E3B44
};

static __device__ void rrounds(uint32_t x[2][2][2][2][2])
{
    int r;
    int j;
    int k;
    int l;
    int m;

//#pragma unroll 16
    for (r = 0;r < CUBEHASH_ROUNDS;++r) {

        /* "add x_0jklm into x_1jklmn modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 7 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS7(x[0][j][k][l][m]);

        /* "swap x_00klm with x_01klm" */
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][0][k][l][m],x[0][1][k][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jk0m with x_1jk1m" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[1][j][k][0][m],x[1][j][k][1][m])

        /* "add x_0jklm into x_1jklm modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 11 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS11(x[0][j][k][l][m]);

        /* "swap x_0j0lm with x_0j1lm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][j][0][l][m],x[0][j][1][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jkl0 with x_1jkl1" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
                    SWAP(x[1][j][k][l][0],x[1][j][k][l][1])

    }
}


static __device__ void block_tox(uint32_t block[16], uint32_t x[2][2][2][2][2])
{
    int k;
    int l;
    int m;
    uint32_t *in = block;

#pragma unroll 2
    for (k = 0;k < 2;++k)
#pragma unroll 2
        for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
                x[0][0][k][l][m] ^= *in++;
}

static __device__ void block_toxC(uint32_t *in, uint32_t x[2][2][2][2][2], int threads, int thread, int offset)
{
    int k;
    int l;
    int m;
    int i;

#pragma unroll 2
    for (k = 0;k < 2;++k)
#pragma unroll 2
        for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m) {
                x[0][0][k][l][m] ^= in[threads*(i+offset)+thread];
                i++;
            }

}

static __device__ void hash_fromx(uint32_t hash[16], uint32_t x[2][2][2][2][2])
{
    int j;
    int k;
    int l;
    int m;
    uint32_t *out = hash;

#pragma unroll 2
    for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    *out++ = x[0][j][k][l][m];
}

static __device__ void hash_fromxC(uint32_t *out, uint32_t x[2][2][2][2][2], int threads, int thread)
{
    int i;
    int j;
    int k;
    int l;
    int m;

#pragma unroll 2
    for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m) {
                    out[i*threads+thread] = x[0][j][k][l][m];
                    i++;
                }
}

void __device__ Init(uint32_t x[2][2][2][2][2])
{
    int i,j,k,l,m;
#if 0
    /* "the first three state words x_00000, x_00001, x_00010" */
    /* "are set to the integers h/8, b, r respectively." */
    /* "the remaining state words are set to 0." */
#pragma unroll 2
    for (i = 0;i < 2;++i)
#pragma unroll 2
      for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
          for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
              x[i][j][k][l][m] = 0;
    x[0][0][0][0][0] = 512/8;
    x[0][0][0][0][1] = CUBEHASH_BLOCKBYTES;
    x[0][0][0][1][0] = CUBEHASH_ROUNDS;

    /* "the state is then transformed invertibly through 10r identical rounds */
    for (i = 0;i < 10;++i) rrounds(x);
#else
    uint32_t *iv = c_IV_512;

#pragma unroll 2
    for (i = 0;i < 2;++i)
#pragma unroll 2
      for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
          for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
              x[i][j][k][l][m] = *iv++;
#endif
}

void __device__ Update32(uint32_t x[2][2][2][2][2], const BitSequence *data)
{
    /* "xor the block into the first b bytes of the state" */
    /* "and then transform the state invertibly through r identical rounds" */
    block_tox((uint32_t*)data, x);
    rrounds(x);
}

void __device__ Update32c(uint32_t x[2][2][2][2][2], const BitSequence *data, int threads, int thread, int offset)
{
    /* "xor the block into the first b bytes of the state" */
    /* "and then transform the state invertibly through r identical rounds" */
    block_toxC((uint32_t*)data, x, threads, thread, offset);
    rrounds(x);
}

void __device__ Final(uint32_t x[2][2][2][2][2], BitSequence *hashval)
{
    int i;

    /* "the integer 1 is xored into the last state word x_11111" */
    x[1][1][1][1][1] ^= 1;

    /* "the state is then transformed invertibly through 10r identical rounds" */
//#pragma unroll 10
    for (i = 0;i < 10;++i) rrounds(x);

    /* "output the first h/8 bytes of the state" */
    hash_fromx((uint32_t*)hashval, x);
}

void __device__ FinalC(uint32_t x[2][2][2][2][2], const BitSequence *hashval, int threads, int thread)
{
    int i;

    /* "the integer 1 is xored into the last state word x_11111" */
    x[1][1][1][1][1] ^= 1;

    /* "the state is then transformed invertibly through 10r identical rounds" */
//#pragma unroll 10
    for (i = 0;i < 10;++i) rrounds(x);

    /* "output the first h/8 bytes of the state" */
    hash_fromxC((uint32_t*)hashval, x, threads, thread);
}


__global__ void x11_cubehash512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[8 * hashPosition];

        uint32_t x[2][2][2][2][2];
        Init(x);

        // erste H�lfte des Hashes (32 bytes)
        Update32(x, (const BitSequence*)Hash);

        // zweite H�lfte des Hashes (32 bytes)
        Update32(x, (const BitSequence*)(Hash+8));

        // Padding Block
        uint32_t last[8];
        last[0] = 0x80;
#pragma unroll 7
        for (int i=1; i < 8; i++) last[i] = 0;
        Update32(x, (const BitSequence*)last);

        Final(x, (BitSequence*)Hash);
    }
}

__global__ void x11_cubehash512c_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        uint32_t x[2][2][2][2][2];
        Init(x);

        // erste H�lfte des Hashes (32 bytes)
        Update32c(x, (const BitSequence*)g_hash, threads, thread, 0);

        // zweite H�lfte des Hashes (32 bytes)
        Update32c(x, (const BitSequence*)g_hash, threads, thread, 8);

        // Padding Block
        uint32_t last[8];
        last[0] = 0x80;
#pragma unroll 7
        for (int i=1; i < 8; i++) last[i] = 0;
        Update32(x, (const BitSequence*)last);

        FinalC(x, (BitSequence*)g_hash, threads, thread);
    }
}


// Setup-Funktionen
__host__ void x11_cubehash512_cpu_init(int thr_id, int threads)
{
    hipMemcpyToSymbol(HIP_SYMBOL( c_IV_512), h_IV_512, sizeof(h_IV_512), 0, hipMemcpyHostToDevice);
}

__host__ void x11_cubehash512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    x11_cubehash512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

//	hipDeviceSynchronize();
}

__host__ void x11_cubehash512c_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    x11_cubehash512c_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

//	hipDeviceSynchronize();
}

