#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"


// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

//#define SHL(x, n)            ((x) << (n))
//#define SHR(x, n)            ((x) >> (n))
#define SHR(x, n) SHR2(x, n) 
#define SHL(x, n) SHL2(x, n) 

#undef ROTL64
#define ROTL64 ROL2


#define CONST_EXP2(i)    q[i+0] + ROTL64(q[i+1], 5)  + q[i+2] + ROTL64(q[i+3], 11) + \
                    q[i+4] + ROTL64(q[i+5], 27) + q[i+6] + SWAPDWORDS2(q[i+7]) + \
                    q[i+8] + ROTL64(q[i+9], 37) + q[i+10] + ROTL64(q[i+11], 43) + \
                    q[i+12] + ROTL64(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])



__device__ __forceinline__ void Compression512(const uint2 *msg, uint2 *hash)
{

	const uint2 precalc[16] =
	{
		{ 0x55555550, 0x55555555 },
		{ 0xAAAAAAA5, 0x5AAAAAAA },
		{ 0xFFFFFFFA, 0x5FFFFFFF },
		{ 0x5555554F, 0x65555555 },
		{ 0xAAAAAAA4, 0x6AAAAAAA },
		{ 0xFFFFFFF9, 0x6FFFFFFF },
		{ 0x5555554E, 0x75555555 },
		{ 0xAAAAAAA3, 0x7AAAAAAA },
		{ 0xFFFFFFF8, 0x7FFFFFFF },
		{ 0x5555554D, 0x85555555 },
		{ 0xAAAAAAA2, 0x8AAAAAAA },
		{ 0xFFFFFFF7, 0x8FFFFFFF },
		{ 0x5555554C, 0x95555555 },
		{ 0xAAAAAAA1, 0x9AAAAAAA },
		{ 0xFFFFFFF6, 0x9FFFFFFF },
		{ 0x5555554B, 0xA5555555 },
	};


	// Compression ref. implementation
	uint2 q[32];
	uint2 tmp;

    tmp = (msg[ 5] ^ hash[ 5]) - (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]) + (msg[14] ^ hash[14]);
    q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[1];
    tmp = (msg[ 6] ^ hash[ 6]) - (msg[ 8] ^ hash[ 8]) + (msg[11] ^ hash[11]) + (msg[14] ^ hash[14]) - (msg[15] ^ hash[15]);
    q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
    tmp = (msg[ 0] ^ hash[ 0]) + (msg[ 7] ^ hash[ 7]) + (msg[ 9] ^ hash[ 9]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 1] ^ hash[ 1]) + (msg[ 8] ^ hash[ 8]) - (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]);
    q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 2] ^ hash[ 2]) + (msg[ 9] ^ hash[ 9]) - (msg[11] ^ hash[11]) - (msg[14] ^ hash[14]);
    q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 2] ^ hash[ 2]) + (msg[10] ^ hash[10]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[6];
    tmp = (msg[ 4] ^ hash[ 4]) - (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) - (msg[11] ^ hash[11]) + (msg[13] ^ hash[13]);
    q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
    tmp = (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 5] ^ hash[ 5]) - (msg[12] ^ hash[12]) - (msg[14] ^ hash[14]);
    q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
    tmp = (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) - (msg[ 6] ^ hash[ 6]) + (msg[13] ^ hash[13]) - (msg[15] ^ hash[15]);
    q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) + (msg[ 6] ^ hash[ 6]) - (msg[ 7] ^ hash[ 7]) + (msg[14] ^ hash[14]);
    q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 7] ^ hash[ 7]) + (msg[15] ^ hash[15]);
    q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[11];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 0] ^ hash[ 0]) - (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) + (msg[ 9] ^ hash[ 9]);
    q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 3] ^ hash[ 3]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[10] ^ hash[10]);
    q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
    tmp = (msg[ 2] ^ hash[ 2]) + (msg[ 4] ^ hash[ 4]) + (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[11] ^ hash[11]);
    q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 5] ^ hash[ 5]) + (msg[ 8] ^ hash[ 8]) - (msg[11] ^ hash[11]) - (msg[12] ^ hash[12]);
    q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
    tmp = (msg[12] ^ hash[12]) - (msg[ 4] ^ hash[ 4]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[13] ^ hash[13]);
    q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

        q[0+16] =
        (SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROTL64(q[0], 13) ^ ROTL64(q[0], 43)) +
        (SHR(q[0+1], 2) ^ SHL(q[0+1], 1) ^ ROTL64(q[0+1], 19) ^ ROTL64(q[0+1], 53)) +
        (SHR(q[0+2], 2) ^ SHL(q[0+2], 2) ^ ROTL64(q[0+2], 28) ^ ROTL64(q[0+2], 59)) +
        (SHR(q[0+3], 1) ^ SHL(q[0+3], 3) ^ ROTL64(q[0+3],  4) ^ ROTL64(q[0+3], 37)) +
        (SHR(q[0+4], 1) ^ SHL(q[0+4], 2) ^ ROTL64(q[0+4], 13) ^ ROTL64(q[0+4], 43)) +
        (SHR(q[0+5], 2) ^ SHL(q[0+5], 1) ^ ROTL64(q[0+5], 19) ^ ROTL64(q[0+5], 53)) +
        (SHR(q[0+6], 2) ^ SHL(q[0+6], 2) ^ ROTL64(q[0+6], 28) ^ ROTL64(q[0+6], 59)) +
        (SHR(q[0+7], 1) ^ SHL(q[0+7], 3) ^ ROTL64(q[0+7],  4) ^ ROTL64(q[0+7], 37)) +
        (SHR(q[0+8], 1) ^ SHL(q[0+8], 2) ^ ROTL64(q[0+8], 13) ^ ROTL64(q[0+8], 43)) +
        (SHR(q[0+9], 2) ^ SHL(q[0+9], 1) ^ ROTL64(q[0+9], 19) ^ ROTL64(q[0+9], 53)) +
        (SHR(q[0+10], 2) ^ SHL(q[0+10], 2) ^ ROTL64(q[0+10], 28) ^ ROTL64(q[0+10], 59)) +
        (SHR(q[0+11], 1) ^ SHL(q[0+11], 3) ^ ROTL64(q[0+11],  4) ^ ROTL64(q[0+11], 37)) +
        (SHR(q[0+12], 1) ^ SHL(q[0+12], 2) ^ ROTL64(q[0+12], 13) ^ ROTL64(q[0+12], 43)) +
        (SHR(q[0+13], 2) ^ SHL(q[0+13], 1) ^ ROTL64(q[0+13], 19) ^ ROTL64(q[0+13], 53)) +
        (SHR(q[0+14], 2) ^ SHL(q[0+14], 2) ^ ROTL64(q[0+14], 28) ^ ROTL64(q[0+14], 59)) +
        (SHR(q[0+15], 1) ^ SHL(q[0+15], 3) ^ ROTL64(q[0+15],  4) ^ ROTL64(q[0+15], 37)) +
		((precalc[0] + ROTL64(msg[0], 0 + 1) +
            ROTL64(msg[0+3], 0+4) - ROTL64(msg[0+10], 0+11) ) ^ hash[0+7]);
		q[1 + 16] =
			(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROTL64(q[1], 13) ^ ROTL64(q[1], 43)) +
			(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROTL64(q[1 + 1], 19) ^ ROTL64(q[1 + 1], 53)) +
			(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROTL64(q[1 + 2], 28) ^ ROTL64(q[1 + 2], 59)) +
			(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROTL64(q[1 + 3], 4) ^ ROTL64(q[1 + 3], 37)) +
			(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROTL64(q[1 + 4], 13) ^ ROTL64(q[1 + 4], 43)) +
			(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROTL64(q[1 + 5], 19) ^ ROTL64(q[1 + 5], 53)) +
			(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROTL64(q[1 + 6], 28) ^ ROTL64(q[1 + 6], 59)) +
			(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROTL64(q[1 + 7], 4) ^ ROTL64(q[1 + 7], 37)) +
			(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROTL64(q[1 + 8], 13) ^ ROTL64(q[1 + 8], 43)) +
			(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROTL64(q[1 + 9], 19) ^ ROTL64(q[1 + 9], 53)) +
			(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROTL64(q[1 + 10], 28) ^ ROTL64(q[1 + 10], 59)) +
			(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROTL64(q[1 + 11], 4) ^ ROTL64(q[1 + 11], 37)) +
			(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROTL64(q[1 + 12], 13) ^ ROTL64(q[1 + 12], 43)) +
			(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROTL64(q[1 + 13], 19) ^ ROTL64(q[1 + 13], 53)) +
			(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROTL64(q[1 + 14], 28) ^ ROTL64(q[1 + 14], 59)) +
			(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROTL64(q[1 + 15], 4) ^ ROTL64(q[1 + 15], 37)) +
			((precalc[1] + ROTL64(msg[1], 1 + 1) +
			ROTL64(msg[1 + 3], 1 + 4) - ROTL64(msg[1 + 10], 1 + 11)) ^ hash[1 + 7]);

		q[2 + 16] = CONST_EXP2(2) +
			((precalc[2] + ROTL64(msg[2], 2 + 1) +
            ROTL64(msg[2+3], 2+4) - ROTL64(msg[2+10], 2+11) ) ^ hash[2+7]);
		q[3 + 16] = CONST_EXP2(3) +
			((precalc[3] + ROTL64(msg[3], 3 + 1) +
			ROTL64(msg[3 + 3], 3 + 4) - ROTL64(msg[3 + 10], 3 + 11)) ^ hash[3 + 7]);
		q[4 + 16] = CONST_EXP2(4) +
			((precalc[4] + ROTL64(msg[4], 4 + 1) +
			ROL8(msg[4 + 3]) - ROTL64(msg[4 + 10], 4 + 11)) ^ hash[4 + 7]);
		q[5 + 16] = CONST_EXP2(5) +
			((precalc[5] + ROTL64(msg[5], 5 + 1) +
			ROTL64(msg[5 + 3], 5 + 4) - ROL16(msg[5 + 10])) ^ hash[5 + 7]);


		q[6 + 16] = CONST_EXP2(6) +
			((precalc[6]+ ROTL64(msg[6], 6 + 1) +
			ROTL64(msg[6 + 3], 6 + 4) - ROTL64(msg[6 - 6], (6 - 6) + 1)) ^ hash[6 + 7]);
		q[7 + 16] = CONST_EXP2(7) +
			((precalc[7] + ROL8(msg[7]) +
			ROTL64(msg[7 + 3], 7 + 4) - ROTL64(msg[7 - 6], (7 - 6) + 1)) ^ hash[7 + 7]);
		q[8 + 16] = CONST_EXP2(8) +
			((precalc[8] + ROTL64(msg[8], 8 + 1) +
			ROTL64(msg[8 + 3], 8 + 4) - ROTL64(msg[8 - 6], (8 - 6) + 1)) ^ hash[8 + 7]);

	q[9 + 16] = CONST_EXP2(9) +
	((precalc[9] + ROTL64(msg[9], 9 + 1) +
		ROTL64(msg[9 + 3], 9 + 4) - ROTL64(msg[9 - 6], (9 - 6) + 1)) ^ hash[9 - 9]);
	q[10 + 16] = CONST_EXP2(10) +
		((precalc[10] + ROTL64(msg[10], 10 + 1) +
		ROTL64(msg[10 + 3], 10 + 4) - ROTL64(msg[10 - 6], (10 - 6) + 1)) ^ hash[10 - 9]);
	q[11 + 16] = CONST_EXP2(11) +
		((precalc[11] + ROTL64(msg[11], 11 + 1) +
		ROTL64(msg[11 + 3], 11 + 4) - ROTL64(msg[11 - 6], (11 - 6) + 1)) ^ hash[11 - 9]);
	q[12 + 16] = CONST_EXP2(12) +
		((precalc[12] + ROTL64(msg[12], 12 + 1) +
		ROL16(msg[12 + 3]) - ROTL64(msg[12 - 6], (12 - 6) + 1)) ^ hash[12 - 9]);

	

		q[13 + 16] = CONST_EXP2(13) +
			((precalc[13] + ROTL64(msg[13], 13 + 1) +
			ROTL64(msg[13 - 13], (13 - 13) + 1) - ROL8(msg[13 - 6])) ^ hash[13 - 9]);
		q[14 + 16] = CONST_EXP2(14) +
			((precalc[14] + ROTL64(msg[14], 14 + 1) +
			ROTL64(msg[14 - 13], (14 - 13) + 1) - ROTL64(msg[14 - 6], (14 - 6) + 1)) ^ hash[14 - 9]);
		q[15 + 16] = CONST_EXP2(15) +
			((precalc[15] + ROL16(msg[15]) +
			ROTL64(msg[15 - 13], (15 - 13) + 1) - ROTL64(msg[15 - 6], (15 - 6) + 1)) ^ hash[15 - 9]);

    uint2 XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

    hash[0] =                       (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[ 0]) + (    XL64    ^ q[24] ^ q[ 0]);
    hash[1] =                       (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[ 1]) + (    XL64    ^ q[25] ^ q[ 1]);
    hash[2] =                       (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[ 2]) + (    XL64    ^ q[26] ^ q[ 2]);
    hash[3] =                       (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[ 3]) + (    XL64    ^ q[27] ^ q[ 3]);
    hash[4] =                       (SHR(XH64, 3) ^     q[20]    ^ msg[ 4]) + (    XL64    ^ q[28] ^ q[ 4]);
    hash[5] =                       (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[ 5]) + (    XL64    ^ q[29] ^ q[ 5]);
    hash[6] =                       (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[ 6]) + (    XL64    ^ q[30] ^ q[ 6]);
    hash[7] =                       (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[ 7]) + (    XL64    ^ q[31] ^ q[ 7]);

    hash[ 8] = ROTL64(hash[4], 9) + (    XH64     ^     q[24]    ^ msg[ 8]) + (SHL(XL64,8) ^ q[23] ^ q[ 8]);
    hash[ 9] = ROTL64(hash[5],10) + (    XH64     ^     q[25]    ^ msg[ 9]) + (SHR(XL64,6) ^ q[16] ^ q[ 9]);
    hash[10] = ROTL64(hash[6],11) + (    XH64     ^     q[26]    ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
    hash[11] = ROTL64(hash[7],12) + (    XH64     ^     q[27]    ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
    hash[12] = ROTL64(hash[0],13) + (    XH64     ^     q[28]    ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
    hash[13] = ROTL64(hash[1],14) + (    XH64     ^     q[29]    ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
    hash[14] = ROTL64(hash[2],15) + (    XH64     ^     q[30]    ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	hash[15] = ROL16(hash[3]) + (XH64     ^     q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);
}
__global__ __launch_bounds__(32, 16)
void quark_bmw512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *const __restrict__ g_hash, const uint32_t *const __restrict__ g_nonceVector)
{
    const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
        const uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        const int hashPosition = nounce - startNounce;
        uint64_t *const inpHash = &g_hash[8 * hashPosition];

		const uint2 hash[16] = 
		{
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};

		const uint64_t hash2[16] =
		{
			 0x8081828384858687 ,
			 0x88898A8B8C8D8E8F ,
			 0x9091929394959697 ,
			 0x98999A9B9C9D9E9F ,
			 0xA0A1A2A3A4A5A6A7 ,
			 0xA8A9AAABACADAEAF ,
			 0xB0B1B2B3B4B5B6B7 ,
			 0xB8B9BABBBCBDBEBF ,
			 0xC0C1C2C3C4C5C6C7 ,
			 0xC8C9CACBCCCDCECF ,
			 0xD0D1D2D3D4D5D6D7 ,
			 0xD8D9DADBDCDDDEDF ,
			 0xE0E1E2E3E4E5E6E7 ,
			 0xE8E9EAEBECEDEEEF ,
			 0xF0F1F2F3F4F5F6F7 ,
			 0xF8F9FAFBFCFDFEFF
		};

		uint2 msg[16];
		uint2 mxh[8];
		uint2 h[16];
		msg[0] = vectorize(inpHash[0]);
		msg[1] = vectorize(inpHash[1]);
		msg[2] = vectorize(inpHash[2]);
		msg[3] = vectorize(inpHash[3]);
		msg[4] = vectorize(inpHash[4]);
		msg[5] = vectorize(inpHash[5]);
		msg[6] = vectorize(inpHash[6]);
		msg[7] = vectorize(inpHash[7]);
		msg[8] = vectorizelow(0x80);
		msg[15] = vectorizelow(512);
		mxh[0] = msg[0] ^ hash[0];
		mxh[1] = msg[1] ^ hash[1];
		mxh[2] = msg[2] ^ hash[2];
		mxh[3] = msg[3] ^ hash[3];
		mxh[4] = msg[4] ^ hash[4];
		mxh[5] = msg[5] ^ hash[5];
		mxh[6] = msg[6] ^ hash[6];
		mxh[7] = msg[7] ^ hash[7];

		const uint2 precalcf[9] =
		{
			{ 0x55555550ul, 0x55555555 },
			{ 0xAAAAAAA5, 0x5AAAAAAA },
			{ 0xFFFFFFFA, 0x5FFFFFFF },
			{ 0x5555554F, 0x65555555 },
			{ 0xAAAAAAA4, 0x6AAAAAAA },
			{ 0xFFFFFFF9, 0x6FFFFFFF },
			{ 0xAAAAAAA1, 0x9AAAAAAA },
			{ 0xFFFFFFF6, 0x9FFFFFFF },
			{ 0x5555554B, 0xA5555555 },
		};

		uint2 q[32];

		uint2 tmp;
		tmp = (mxh[5]) - (mxh[7]) + vectorize(hash2[10] + hash2[13] + hash2[14]);
		q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[1];
		tmp = (mxh[6]) + vectorize(hash2[11] + hash2[14] - (512 ^ hash2[15]) - (0x80^hash2[8]));
		q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
		tmp = (mxh[0] + mxh[7]) + vectorize(hash2[9] - hash2[12] + (512 ^ hash2[15]));
		q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
		tmp = (mxh[0] - mxh[1]) + vectorize((0x80 ^ hash2[8])- hash2[10] + hash2[13]);
		q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
		tmp = (mxh[1] + mxh[2]) + vectorize(hash2[9] - hash2[11] - hash2[14]);
		q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
		tmp = (mxh[3] - mxh[2]) + vectorize(hash2[10] - hash2[12] + (512 ^ hash2[15]));
		q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[6];
		tmp = (mxh[4]) - (mxh[0]) - (mxh[3]) + vectorize(hash2[13] - hash2[11] );
		q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
		tmp = (mxh[1]) - (mxh[4]) - (mxh[5]) + vectorize(-hash2[12] - hash2[14]);
		q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
		tmp = (mxh[2]) - (mxh[5]) - (mxh[6]) + vectorize(hash2[13] - (512 ^ hash2[15]));
		q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
		tmp = (mxh[0]) - (mxh[3]) + (mxh[6]) - (mxh[7]) + (hash[14]);
		q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
		tmp = vectorize((512 ^ hash2[15]) + (0x80 ^ hash2[8])) - (mxh[1]) - (mxh[4]) - (mxh[7]);
		q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[11];
		tmp = vectorize(hash2[9] + (0x80 ^ hash2[8])) - (mxh[0]) - (mxh[2]) - (mxh[5]);
		q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
		tmp = (mxh[1]) + (mxh[3]) - (mxh[6]) + vectorize(hash2[10] - hash2[9]) ;
		q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
		tmp = (mxh[2]) + (mxh[4]) + (mxh[7]) + vectorize(hash2[10] + hash2[11]);
		q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
		tmp = (mxh[3]) - (mxh[5]) + vectorize((0x80 ^ hash2[8]) - hash2[11] - hash2[12]);
		q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
		tmp = vectorize(hash2[12] - hash2[9] + hash2[13]) - (mxh[4]) - (mxh[6]);
		q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

		q[0 + 16] =
			(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROTL64(q[0], 13) ^ ROTL64(q[0], 43)) +
			(SHR(q[0 + 1], 2) ^ SHL(q[0 + 1], 1) ^ ROTL64(q[0 + 1], 19) ^ ROTL64(q[0 + 1], 53)) +
			(SHR(q[0 + 2], 2) ^ SHL(q[0 + 2], 2) ^ ROTL64(q[0 + 2], 28) ^ ROTL64(q[0 + 2], 59)) +
			(SHR(q[0 + 3], 1) ^ SHL(q[0 + 3], 3) ^ ROTL64(q[0 + 3], 4) ^ ROTL64(q[0 + 3], 37)) +
			(SHR(q[0 + 4], 1) ^ SHL(q[0 + 4], 2) ^ ROTL64(q[0 + 4], 13) ^ ROTL64(q[0 + 4], 43)) +
			(SHR(q[0 + 5], 2) ^ SHL(q[0 + 5], 1) ^ ROTL64(q[0 + 5], 19) ^ ROTL64(q[0 + 5], 53)) +
			(SHR(q[0 + 6], 2) ^ SHL(q[0 + 6], 2) ^ ROTL64(q[0 + 6], 28) ^ ROTL64(q[0 + 6], 59)) +
			(SHR(q[0 + 7], 1) ^ SHL(q[0 + 7], 3) ^ ROTL64(q[0 + 7], 4) ^ ROTL64(q[0 + 7], 37)) +
			(SHR(q[0 + 8], 1) ^ SHL(q[0 + 8], 2) ^ ROTL64(q[0 + 8], 13) ^ ROTL64(q[0 + 8], 43)) +
			(SHR(q[0 + 9], 2) ^ SHL(q[0 + 9], 1) ^ ROTL64(q[0 + 9], 19) ^ ROTL64(q[0 + 9], 53)) +
			(SHR(q[0 + 10], 2) ^ SHL(q[0 + 10], 2) ^ ROTL64(q[0 + 10], 28) ^ ROTL64(q[0 + 10], 59)) +
			(SHR(q[0 + 11], 1) ^ SHL(q[0 + 11], 3) ^ ROTL64(q[0 + 11], 4) ^ ROTL64(q[0 + 11], 37)) +
			(SHR(q[0 + 12], 1) ^ SHL(q[0 + 12], 2) ^ ROTL64(q[0 + 12], 13) ^ ROTL64(q[0 + 12], 43)) +
			(SHR(q[0 + 13], 2) ^ SHL(q[0 + 13], 1) ^ ROTL64(q[0 + 13], 19) ^ ROTL64(q[0 + 13], 53)) +
			(SHR(q[0 + 14], 2) ^ SHL(q[0 + 14], 2) ^ ROTL64(q[0 + 14], 28) ^ ROTL64(q[0 + 14], 59)) +
			(SHR(q[0 + 15], 1) ^ SHL(q[0 + 15], 3) ^ ROTL64(q[0 + 15], 4) ^ ROTL64(q[0 + 15], 37)) +
			((precalcf[0] + ROTL64(msg[0], 0 + 1) +
			ROTL64(msg[0 + 3], 0 + 4)) ^ hash[0 + 7]);
		q[1 + 16] =
			(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROTL64(q[1], 13) ^ ROTL64(q[1], 43)) +
			(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROTL64(q[1 + 1], 19) ^ ROTL64(q[1 + 1], 53)) +
			(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROTL64(q[1 + 2], 28) ^ ROTL64(q[1 + 2], 59)) +
			(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROTL64(q[1 + 3], 4) ^ ROTL64(q[1 + 3], 37)) +
			(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROTL64(q[1 + 4], 13) ^ ROTL64(q[1 + 4], 43)) +
			(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROTL64(q[1 + 5], 19) ^ ROTL64(q[1 + 5], 53)) +
			(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROTL64(q[1 + 6], 28) ^ ROTL64(q[1 + 6], 59)) +
			(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROTL64(q[1 + 7], 4) ^ ROTL64(q[1 + 7], 37)) +
			(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROTL64(q[1 + 8], 13) ^ ROTL64(q[1 + 8], 43)) +
			(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROTL64(q[1 + 9], 19) ^ ROTL64(q[1 + 9], 53)) +
			(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROTL64(q[1 + 10], 28) ^ ROTL64(q[1 + 10], 59)) +
			(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROTL64(q[1 + 11], 4) ^ ROTL64(q[1 + 11], 37)) +
			(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROTL64(q[1 + 12], 13) ^ ROTL64(q[1 + 12], 43)) +
			(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROTL64(q[1 + 13], 19) ^ ROTL64(q[1 + 13], 53)) +
			(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROTL64(q[1 + 14], 28) ^ ROTL64(q[1 + 14], 59)) +
			(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROTL64(q[1 + 15], 4) ^ ROTL64(q[1 + 15], 37)) +
			((precalcf[1] + ROTL64(msg[1], 1 + 1) +
			ROTL64(msg[1 + 3], 1 + 4)) ^ hash[1 + 7]);

		q[2 + 16] = CONST_EXP2(2) +
			((precalcf[2] + ROTL64(msg[2], 2 + 1) +
			ROTL64(msg[2 + 3], 2 + 4)) ^ hash[2 + 7]);
		q[3 + 16] = CONST_EXP2(3) +
			((precalcf[3] + ROTL64(msg[3], 3 + 1) +
			ROTL64(msg[3 + 3], 3 + 4)) ^ hash[3 + 7]);
		q[4 + 16] = CONST_EXP2(4) +
			((precalcf[4] + ROTL64(msg[4], 4 + 1) +
			ROL8(msg[4 + 3])) ^ hash[4 + 7]);
		q[5 + 16] = CONST_EXP2(5) +
			((precalcf[5] + ROTL64(msg[5], 5 + 1) +
			ROTL64(msg[5 + 3], 5 + 4) - ROL16(msg[5 + 10])) ^ hash[5 + 7]);


		//#pragma unroll 3
		for (int i = 6; i < 9; i++) {
			q[i + 16] = CONST_EXP2(i) +
				((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) -
				ROTL64(msg[i - 6], (i - 6) + 1)) ^ hash[i + 7]);
		}

		q[25] = CONST_EXP2(9) +
			((vectorize((25)*(0x0555555555555555ull)) - ROTL64(msg[3], 4)) ^ hash[0]);
		q[26] = CONST_EXP2(10) +
			((vectorize((26)*(0x0555555555555555ull)) - ROTL64(msg[4], 5)) ^ hash[1]);
		q[27] = CONST_EXP2(11) +
			((vectorize((27)*(0x0555555555555555ull)) - ROTL64(msg[5], 6)) ^ hash[2]);
		q[28] = CONST_EXP2(12) +
			((vectorize((28)*(0x0555555555555555ull)) +	ROL16(msg[15]) - ROTL64(msg[6], 7)) ^ hash[3]);

		q[13 + 16] = CONST_EXP2(13) +
			((precalcf[6] + 
			ROTL64(msg[13 - 13], (13 - 13) + 1) - ROL8(msg[13 - 6])) ^ hash[13 - 9]);
		q[14 + 16] = CONST_EXP2(14) +
			((precalcf[7] + 
			ROTL64(msg[14 - 13], (14 - 13) + 1) - ROTL64(msg[14 - 6], (14 - 6) + 1)) ^ hash[14 - 9]);
		q[15 + 16] = CONST_EXP2(15) +
			((precalcf[8] + ROL16(msg[15]) +
			ROTL64(msg[15 - 13], (15 - 13) + 1)) ^ hash[15 - 9]);

		uint2 XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
		uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

		h[0] = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ msg[0]) + (XL64    ^ q[24] ^ q[0]);
		h[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ msg[1]) + (XL64    ^ q[25] ^ q[1]);
		h[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ msg[2]) + (XL64    ^ q[26] ^ q[2]);
		h[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ msg[3]) + (XL64    ^ q[27] ^ q[3]);
		h[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + (XL64    ^ q[28] ^ q[4]);
		h[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ msg[5]) + (XL64    ^ q[29] ^ q[5]);
		h[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ msg[6]) + (XL64    ^ q[30] ^ q[6]);
		h[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ msg[7]) + (XL64    ^ q[31] ^ q[7]);

		h[8] = ROTL64(h[4], 9) + (XH64     ^     q[24] ^ msg[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);
		h[9] = ROTL64(h[5], 10) + (XH64     ^     q[25]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
		h[10] = ROTL64(h[6], 11) + (XH64     ^     q[26]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
		h[11] = ROTL64(h[7], 12) + (XH64     ^     q[27]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
		h[12] = ROTL64(h[0], 13) + (XH64     ^     q[28]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
		h[13] = ROTL64(h[1], 14) + (XH64     ^     q[29]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
		h[14] = ROTL64(h[2], 15) + (XH64     ^     q[30]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
		h[15] = ROL16(h[3]) + (XH64     ^     q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);

		const uint2 cmsg[16] =
		{
			0xaaaaaaa0, 0xaaaaaaaa,
			0xaaaaaaa1, 0xaaaaaaaa,
			0xaaaaaaa2, 0xaaaaaaaa,
			0xaaaaaaa3, 0xaaaaaaaa,
			0xaaaaaaa4, 0xaaaaaaaa,
			0xaaaaaaa5, 0xaaaaaaaa,
			0xaaaaaaa6, 0xaaaaaaaa,
			0xaaaaaaa7, 0xaaaaaaaa,
			0xaaaaaaa8, 0xaaaaaaaa,
			0xaaaaaaa9, 0xaaaaaaaa,
			0xaaaaaaaa, 0xaaaaaaaa,
			0xaaaaaaab, 0xaaaaaaaa,
			0xaaaaaaac, 0xaaaaaaaa,
			0xaaaaaaad, 0xaaaaaaaa,
			0xaaaaaaae, 0xaaaaaaaa,
			0xaaaaaaaf, 0xaaaaaaaa
		};

#pragma unroll 16
		for (int i = 0; i < 16; i++)
		{
			msg[i] = cmsg[i] ^ h[i];
		}


		const uint2 precalc[16] =
		{
			{ 0x55555550, 0x55555555 },
			{ 0xAAAAAAA5, 0x5AAAAAAA },
			{ 0xFFFFFFFA, 0x5FFFFFFF },
			{ 0x5555554F, 0x65555555 },
			{ 0xAAAAAAA4, 0x6AAAAAAA },
			{ 0xFFFFFFF9, 0x6FFFFFFF },
			{ 0x5555554E, 0x75555555 },
			{ 0xAAAAAAA3, 0x7AAAAAAA },
			{ 0xFFFFFFF8, 0x7FFFFFFF },
			{ 0x5555554D, 0x85555555 },
			{ 0xAAAAAAA2, 0x8AAAAAAA },
			{ 0xFFFFFFF7, 0x8FFFFFFF },
			{ 0x5555554C, 0x95555555 },
			{ 0xAAAAAAA1, 0x9AAAAAAA },
			{ 0xFFFFFFF6, 0x9FFFFFFF },
			{ 0x5555554B, 0xA5555555 },
		};

		tmp = (msg[5]) - (msg[7]) + (msg[10]) + (msg[13]) + (msg[14]);
		q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[1];
		tmp = (msg[6]) - (msg[8]) + (msg[11]) + (msg[14]) - (msg[15]);
		q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[2];
		tmp = (msg[0]) + (msg[7]) + (msg[9]) - (msg[12]) + (msg[15]);
		q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[3];
		tmp = (msg[0]) - (msg[1]) + (msg[8]) - (msg[10]) + (msg[13]);
		q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[4];
		tmp = (msg[1]) + (msg[2]) + (msg[9]) - (msg[11]) - (msg[14]);
		q[4] = (SHR(tmp, 1) ^ tmp) + cmsg[5];
		tmp = (msg[3]) - (msg[2]) + (msg[10]) - (msg[12]) + (msg[15]);
		q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[6];
		tmp = (msg[4]) - (msg[0]) - (msg[3]) - (msg[11]) + (msg[13]);
		q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[7];
		tmp = (msg[1]) - (msg[4]) - (msg[5]) - (msg[12]) - (msg[14]);
		q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[8];
		tmp = (msg[2]) - (msg[5]) - (msg[6]) + (msg[13]) - (msg[15]);
		q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[9];
		tmp = (msg[0]) - (msg[3]) + (msg[6]) - (msg[7]) + (msg[14]);
		q[9] = (SHR(tmp, 1) ^ tmp) + cmsg[10];
		tmp = (msg[8]) - (msg[1]) - (msg[4]) - (msg[7]) + (msg[15]);
		q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[11];
		tmp = (msg[8]) - (msg[0]) - (msg[2]) - (msg[5]) + (msg[9]);
		q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[12];
		tmp = (msg[1]) + (msg[3]) - (msg[6]) - (msg[9]) + (msg[10]);
		q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[13];
		tmp = (msg[2]) + (msg[4]) + (msg[7]) + (msg[10]) + (msg[11]);
		q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[14];
		tmp = (msg[3]) - (msg[5]) + (msg[8]) - (msg[11]) - (msg[12]);
		q[14] = (SHR(tmp, 1) ^ tmp) + cmsg[15];
		tmp = (msg[12]) - (msg[4]) - (msg[6]) - (msg[9]) + (msg[13]);
		q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[0];

		q[0 + 16] =
			(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROTL64(q[0], 13) ^ ROTL64(q[0], 43)) +
			(SHR(q[0 + 1], 2) ^ SHL(q[0 + 1], 1) ^ ROTL64(q[0 + 1], 19) ^ ROTL64(q[0 + 1], 53)) +
			(SHR(q[0 + 2], 2) ^ SHL(q[0 + 2], 2) ^ ROTL64(q[0 + 2], 28) ^ ROTL64(q[0 + 2], 59)) +
			(SHR(q[0 + 3], 1) ^ SHL(q[0 + 3], 3) ^ ROTL64(q[0 + 3], 4) ^ ROTL64(q[0 + 3], 37)) +
			(SHR(q[0 + 4], 1) ^ SHL(q[0 + 4], 2) ^ ROTL64(q[0 + 4], 13) ^ ROTL64(q[0 + 4], 43)) +
			(SHR(q[0 + 5], 2) ^ SHL(q[0 + 5], 1) ^ ROTL64(q[0 + 5], 19) ^ ROTL64(q[0 + 5], 53)) +
			(SHR(q[0 + 6], 2) ^ SHL(q[0 + 6], 2) ^ ROTL64(q[0 + 6], 28) ^ ROTL64(q[0 + 6], 59)) +
			(SHR(q[0 + 7], 1) ^ SHL(q[0 + 7], 3) ^ ROTL64(q[0 + 7], 4) ^ ROTL64(q[0 + 7], 37)) +
			(SHR(q[0 + 8], 1) ^ SHL(q[0 + 8], 2) ^ ROTL64(q[0 + 8], 13) ^ ROTL64(q[0 + 8], 43)) +
			(SHR(q[0 + 9], 2) ^ SHL(q[0 + 9], 1) ^ ROTL64(q[0 + 9], 19) ^ ROTL64(q[0 + 9], 53)) +
			(SHR(q[0 + 10], 2) ^ SHL(q[0 + 10], 2) ^ ROTL64(q[0 + 10], 28) ^ ROTL64(q[0 + 10], 59)) +
			(SHR(q[0 + 11], 1) ^ SHL(q[0 + 11], 3) ^ ROTL64(q[0 + 11], 4) ^ ROTL64(q[0 + 11], 37)) +
			(SHR(q[0 + 12], 1) ^ SHL(q[0 + 12], 2) ^ ROTL64(q[0 + 12], 13) ^ ROTL64(q[0 + 12], 43)) +
			(SHR(q[0 + 13], 2) ^ SHL(q[0 + 13], 1) ^ ROTL64(q[0 + 13], 19) ^ ROTL64(q[0 + 13], 53)) +
			(SHR(q[0 + 14], 2) ^ SHL(q[0 + 14], 2) ^ ROTL64(q[0 + 14], 28) ^ ROTL64(q[0 + 14], 59)) +
			(SHR(q[0 + 15], 1) ^ SHL(q[0 + 15], 3) ^ ROTL64(q[0 + 15], 4) ^ ROTL64(q[0 + 15], 37)) +
			((precalc[0] + ROTL64(h[0], 0 + 1) +
			ROTL64(h[0 + 3], 0 + 4) - ROTL64(h[0 + 10], 0 + 11)) ^ cmsg[0 + 7]);
		q[1 + 16] =
			(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROTL64(q[1], 13) ^ ROTL64(q[1], 43)) +
			(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROTL64(q[1 + 1], 19) ^ ROTL64(q[1 + 1], 53)) +
			(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROTL64(q[1 + 2], 28) ^ ROTL64(q[1 + 2], 59)) +
			(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROTL64(q[1 + 3], 4) ^ ROTL64(q[1 + 3], 37)) +
			(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROTL64(q[1 + 4], 13) ^ ROTL64(q[1 + 4], 43)) +
			(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROTL64(q[1 + 5], 19) ^ ROTL64(q[1 + 5], 53)) +
			(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROTL64(q[1 + 6], 28) ^ ROTL64(q[1 + 6], 59)) +
			(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROTL64(q[1 + 7], 4) ^ ROTL64(q[1 + 7], 37)) +
			(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROTL64(q[1 + 8], 13) ^ ROTL64(q[1 + 8], 43)) +
			(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROTL64(q[1 + 9], 19) ^ ROTL64(q[1 + 9], 53)) +
			(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROTL64(q[1 + 10], 28) ^ ROTL64(q[1 + 10], 59)) +
			(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROTL64(q[1 + 11], 4) ^ ROTL64(q[1 + 11], 37)) +
			(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROTL64(q[1 + 12], 13) ^ ROTL64(q[1 + 12], 43)) +
			(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROTL64(q[1 + 13], 19) ^ ROTL64(q[1 + 13], 53)) +
			(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROTL64(q[1 + 14], 28) ^ ROTL64(q[1 + 14], 59)) +
			(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROTL64(q[1 + 15], 4) ^ ROTL64(q[1 + 15], 37)) +
			((precalc[1] + ROTL64(h[1], 1 + 1) +
			ROTL64(h[1 + 3], 1 + 4) - ROTL64(h[1 + 10], 1 + 11)) ^ cmsg[1 + 7]);

		q[2 + 16] = CONST_EXP2(2) +
			((precalc[2] + ROTL64(h[2], 2 + 1) +
			ROTL64(h[2 + 3], 2 + 4) - ROTL64(h[2 + 10], 2 + 11)) ^ cmsg[2 + 7]);
		q[3 + 16] = CONST_EXP2(3) +
			((precalc[3] + ROTL64(h[3], 3 + 1) +
			ROTL64(h[3 + 3], 3 + 4) - ROTL64(h[3 + 10], 3 + 11)) ^ cmsg[3 + 7]);
		q[4 + 16] = CONST_EXP2(4) +
			((precalc[4] + ROTL64(h[4], 4 + 1) +
			ROL8(h[4 + 3]) - ROTL64(h[4 + 10], 4 + 11)) ^ cmsg[4 + 7]);
		q[5 + 16] = CONST_EXP2(5) +
			((precalc[5] + ROTL64(h[5], 5 + 1) +
			ROTL64(h[5 + 3], 5 + 4) - ROL16(h[5 + 10])) ^ cmsg[5 + 7]);


		q[6 + 16] = CONST_EXP2(6) +
			((precalc[6] + ROTL64(h[6], 6 + 1) +
			ROTL64(h[6 + 3], 6 + 4) - ROTL64(h[6 - 6], (6 - 6) + 1)) ^ cmsg[6 + 7]);
		q[7 + 16] = CONST_EXP2(7) +
			((precalc[7] + ROL8(h[7]) +
			ROTL64(h[7 + 3], 7 + 4) - ROTL64(h[7 - 6], (7 - 6) + 1)) ^ cmsg[7 + 7]);
		q[8 + 16] = CONST_EXP2(8) +
			((precalc[8] + ROTL64(h[8], 8 + 1) +
			ROTL64(h[8 + 3], 8 + 4) - ROTL64(h[8 - 6], (8 - 6) + 1)) ^ cmsg[8 + 7]);

		q[9 + 16] = CONST_EXP2(9) +
			((precalc[9] + ROTL64(h[9], 9 + 1) +
			ROTL64(h[9 + 3], 9 + 4) - ROTL64(h[9 - 6], (9 - 6) + 1)) ^ cmsg[9 - 9]);
		q[10 + 16] = CONST_EXP2(10) +
			((precalc[10] + ROTL64(h[10], 10 + 1) +
			ROTL64(h[10 + 3], 10 + 4) - ROTL64(h[10 - 6], (10 - 6) + 1)) ^ cmsg[10 - 9]);
		q[11 + 16] = CONST_EXP2(11) +
			((precalc[11] + ROTL64(h[11], 11 + 1) +
			ROTL64(h[11 + 3], 11 + 4) - ROTL64(h[11 - 6], (11 - 6) + 1)) ^ cmsg[11 - 9]);
		q[12 + 16] = CONST_EXP2(12) +
			((precalc[12] + ROTL64(h[12], 12 + 1) +
			ROL16(h[12 + 3]) - ROTL64(h[12 - 6], (12 - 6) + 1)) ^ cmsg[12 - 9]);



		q[13 + 16] = CONST_EXP2(13) +
			((precalc[13] + ROTL64(h[13], 13 + 1) +
			ROTL64(h[13 - 13], (13 - 13) + 1) - ROL8(h[13 - 6])) ^ cmsg[13 - 9]);
		q[14 + 16] = CONST_EXP2(14) +
			((precalc[14] + ROTL64(h[14], 14 + 1) +
			ROTL64(h[14 - 13], (14 - 13) + 1) - ROTL64(h[14 - 6], (14 - 6) + 1)) ^ cmsg[14 - 9]);
		q[15 + 16] = CONST_EXP2(15) +
			((precalc[15] + ROL16(h[15]) +
			ROTL64(h[15 - 13], (15 - 13) + 1) - ROTL64(h[15 - 6], (15 - 6) + 1)) ^ cmsg[15 - 9]);

		XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
		XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

		msg[0] = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ h[0]) + (XL64    ^ q[24] ^ q[0]);
		msg[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ h[1]) + (XL64    ^ q[25] ^ q[1]);
		msg[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ h[2]) + (XL64    ^ q[26] ^ q[2]);
		msg[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ h[3]) + (XL64    ^ q[27] ^ q[3]);
		msg[4] = (SHR(XH64, 3) ^ q[20] ^ h[4]) + (XL64    ^ q[28] ^ q[4]);
		msg[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ h[5]) + (XL64    ^ q[29] ^ q[5]);
		msg[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ h[6]) + (XL64    ^ q[30] ^ q[6]);
		msg[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ h[7]) + (XL64    ^ q[31] ^ q[7]);
		msg[8] = ROTL64(msg[4], 9) + (XH64     ^     q[24] ^ h[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);

		msg[9] = ROTL64(msg[5], 10) + (XH64     ^     q[25] ^ h[9]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
		msg[10] = ROTL64(msg[6], 11) + (XH64     ^     q[26] ^ h[10]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
		msg[11] = ROTL64(msg[7], 12) + (XH64     ^     q[27] ^ h[11]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
		msg[12] = ROTL64(msg[0], 13) + (XH64     ^     q[28] ^ h[12]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
		msg[13] = ROTL64(msg[1], 14) + (XH64     ^     q[29] ^ h[13]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
		msg[14] = ROTL64(msg[2], 15) + (XH64     ^     q[30] ^ h[14]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
		msg[15] = ROL16(msg[3]) + (XH64     ^     q[31] ^ h[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);

		inpHash[0] = devectorize(msg[0 + 8]);
		inpHash[1] = devectorize(msg[1 + 8]);
		inpHash[2] = devectorize(msg[2 + 8]);
		inpHash[3] = devectorize(msg[3 + 8]);
		inpHash[4] = devectorize(msg[4 + 8]);
		inpHash[5] = devectorize(msg[5 + 8]);
		inpHash[6] = devectorize(msg[6 + 8]);
		inpHash[7] = devectorize(msg[7 + 8]);
	}
}

__global__ __launch_bounds__(32, 16)
void quark_bmw512_gpu_hash_64_quark(uint32_t threads, uint32_t startNounce, uint64_t *const __restrict__ g_hash, uint32_t *g_nonceVector)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (startNounce + thread);

		const int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		const uint2 hash[16] =
		{
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};

		const uint64_t hash2[16] =
		{
			0x8081828384858687,
			0x88898A8B8C8D8E8F,
			0x9091929394959697,
			0x98999A9B9C9D9E9F,
			0xA0A1A2A3A4A5A6A7,
			0xA8A9AAABACADAEAF,
			0xB0B1B2B3B4B5B6B7,
			0xB8B9BABBBCBDBEBF,
			0xC0C1C2C3C4C5C6C7,
			0xC8C9CACBCCCDCECF,
			0xD0D1D2D3D4D5D6D7,
			0xD8D9DADBDCDDDEDF,
			0xE0E1E2E3E4E5E6E7,
			0xE8E9EAEBECEDEEEF,
			0xF0F1F2F3F4F5F6F7,
			0xF8F9FAFBFCFDFEFF
		};

		uint2 msg[16];
		uint2 mxh[8];
		uint2 h[16];
		msg[0] = vectorize(inpHash[0]);
		msg[1] = vectorize(inpHash[1]);
		msg[2] = vectorize(inpHash[2]);
		msg[3] = vectorize(inpHash[3]);
		msg[4] = vectorize(inpHash[4]);
		msg[5] = vectorize(inpHash[5]);
		msg[6] = vectorize(inpHash[6]);
		msg[7] = vectorize(inpHash[7]);
		msg[8] = vectorizelow(0x80);
		msg[15] = vectorizelow(512);
		mxh[0] = msg[0] ^ hash[0];
		mxh[1] = msg[1] ^ hash[1];
		mxh[2] = msg[2] ^ hash[2];
		mxh[3] = msg[3] ^ hash[3];
		mxh[4] = msg[4] ^ hash[4];
		mxh[5] = msg[5] ^ hash[5];
		mxh[6] = msg[6] ^ hash[6];
		mxh[7] = msg[7] ^ hash[7];

		const uint2 precalcf[9] =
		{
			{ 0x55555550ul, 0x55555555 },
			{ 0xAAAAAAA5, 0x5AAAAAAA },
			{ 0xFFFFFFFA, 0x5FFFFFFF },
			{ 0x5555554F, 0x65555555 },
			{ 0xAAAAAAA4, 0x6AAAAAAA },
			{ 0xFFFFFFF9, 0x6FFFFFFF },
			{ 0xAAAAAAA1, 0x9AAAAAAA },
			{ 0xFFFFFFF6, 0x9FFFFFFF },
			{ 0x5555554B, 0xA5555555 },
		};

		uint2 q[32];

		uint2 tmp;
		tmp = (mxh[5]) - (mxh[7]) + vectorize(hash2[10] + hash2[13] + hash2[14]);
		q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[1];
		tmp = (mxh[6]) + vectorize(hash2[11] + hash2[14] - (512 ^ hash2[15]) - (0x80 ^ hash2[8]));
		q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
		tmp = (mxh[0] + mxh[7]) + vectorize(hash2[9] - hash2[12] + (512 ^ hash2[15]));
		q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
		tmp = (mxh[0] - mxh[1]) + vectorize((0x80 ^ hash2[8]) - hash2[10] + hash2[13]);
		q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
		tmp = (mxh[1] + mxh[2]) + vectorize(hash2[9] - hash2[11] - hash2[14]);
		q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
		tmp = (mxh[3] - mxh[2]) + vectorize(hash2[10] - hash2[12] + (512 ^ hash2[15]));
		q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[6];
		tmp = (mxh[4]) - (mxh[0]) - (mxh[3]) + vectorize(hash2[13] - hash2[11]);
		q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
		tmp = (mxh[1]) - (mxh[4]) - (mxh[5]) + vectorize(-hash2[12] - hash2[14]);
		q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
		tmp = (mxh[2]) - (mxh[5]) - (mxh[6]) + vectorize(hash2[13] - (512 ^ hash2[15]));
		q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
		tmp = (mxh[0]) - (mxh[3]) + (mxh[6]) - (mxh[7]) + (hash[14]);
		q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
		tmp = vectorize((512 ^ hash2[15]) + (0x80 ^ hash2[8])) - (mxh[1]) - (mxh[4]) - (mxh[7]);
		q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[11];
		tmp = vectorize(hash2[9] + (0x80 ^ hash2[8])) - (mxh[0]) - (mxh[2]) - (mxh[5]);
		q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
		tmp = (mxh[1]) + (mxh[3]) - (mxh[6]) + vectorize(hash2[10] - hash2[9]);
		q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
		tmp = (mxh[2]) + (mxh[4]) + (mxh[7]) + vectorize(hash2[10] + hash2[11]);
		q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
		tmp = (mxh[3]) - (mxh[5]) + vectorize((0x80 ^ hash2[8]) - hash2[11] - hash2[12]);
		q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
		tmp = vectorize(hash2[12] - hash2[9] + hash2[13]) - (mxh[4]) - (mxh[6]);
		q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

		q[0 + 16] =
			(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROTL64(q[0], 13) ^ ROTL64(q[0], 43)) +
			(SHR(q[0 + 1], 2) ^ SHL(q[0 + 1], 1) ^ ROTL64(q[0 + 1], 19) ^ ROTL64(q[0 + 1], 53)) +
			(SHR(q[0 + 2], 2) ^ SHL(q[0 + 2], 2) ^ ROTL64(q[0 + 2], 28) ^ ROTL64(q[0 + 2], 59)) +
			(SHR(q[0 + 3], 1) ^ SHL(q[0 + 3], 3) ^ ROTL64(q[0 + 3], 4) ^ ROTL64(q[0 + 3], 37)) +
			(SHR(q[0 + 4], 1) ^ SHL(q[0 + 4], 2) ^ ROTL64(q[0 + 4], 13) ^ ROTL64(q[0 + 4], 43)) +
			(SHR(q[0 + 5], 2) ^ SHL(q[0 + 5], 1) ^ ROTL64(q[0 + 5], 19) ^ ROTL64(q[0 + 5], 53)) +
			(SHR(q[0 + 6], 2) ^ SHL(q[0 + 6], 2) ^ ROTL64(q[0 + 6], 28) ^ ROTL64(q[0 + 6], 59)) +
			(SHR(q[0 + 7], 1) ^ SHL(q[0 + 7], 3) ^ ROTL64(q[0 + 7], 4) ^ ROTL64(q[0 + 7], 37)) +
			(SHR(q[0 + 8], 1) ^ SHL(q[0 + 8], 2) ^ ROTL64(q[0 + 8], 13) ^ ROTL64(q[0 + 8], 43)) +
			(SHR(q[0 + 9], 2) ^ SHL(q[0 + 9], 1) ^ ROTL64(q[0 + 9], 19) ^ ROTL64(q[0 + 9], 53)) +
			(SHR(q[0 + 10], 2) ^ SHL(q[0 + 10], 2) ^ ROTL64(q[0 + 10], 28) ^ ROTL64(q[0 + 10], 59)) +
			(SHR(q[0 + 11], 1) ^ SHL(q[0 + 11], 3) ^ ROTL64(q[0 + 11], 4) ^ ROTL64(q[0 + 11], 37)) +
			(SHR(q[0 + 12], 1) ^ SHL(q[0 + 12], 2) ^ ROTL64(q[0 + 12], 13) ^ ROTL64(q[0 + 12], 43)) +
			(SHR(q[0 + 13], 2) ^ SHL(q[0 + 13], 1) ^ ROTL64(q[0 + 13], 19) ^ ROTL64(q[0 + 13], 53)) +
			(SHR(q[0 + 14], 2) ^ SHL(q[0 + 14], 2) ^ ROTL64(q[0 + 14], 28) ^ ROTL64(q[0 + 14], 59)) +
			(SHR(q[0 + 15], 1) ^ SHL(q[0 + 15], 3) ^ ROTL64(q[0 + 15], 4) ^ ROTL64(q[0 + 15], 37)) +
			((precalcf[0] + ROTL64(msg[0], 0 + 1) +
			ROTL64(msg[0 + 3], 0 + 4)) ^ hash[0 + 7]);
		q[1 + 16] =
			(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROTL64(q[1], 13) ^ ROTL64(q[1], 43)) +
			(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROTL64(q[1 + 1], 19) ^ ROTL64(q[1 + 1], 53)) +
			(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROTL64(q[1 + 2], 28) ^ ROTL64(q[1 + 2], 59)) +
			(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROTL64(q[1 + 3], 4) ^ ROTL64(q[1 + 3], 37)) +
			(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROTL64(q[1 + 4], 13) ^ ROTL64(q[1 + 4], 43)) +
			(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROTL64(q[1 + 5], 19) ^ ROTL64(q[1 + 5], 53)) +
			(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROTL64(q[1 + 6], 28) ^ ROTL64(q[1 + 6], 59)) +
			(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROTL64(q[1 + 7], 4) ^ ROTL64(q[1 + 7], 37)) +
			(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROTL64(q[1 + 8], 13) ^ ROTL64(q[1 + 8], 43)) +
			(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROTL64(q[1 + 9], 19) ^ ROTL64(q[1 + 9], 53)) +
			(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROTL64(q[1 + 10], 28) ^ ROTL64(q[1 + 10], 59)) +
			(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROTL64(q[1 + 11], 4) ^ ROTL64(q[1 + 11], 37)) +
			(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROTL64(q[1 + 12], 13) ^ ROTL64(q[1 + 12], 43)) +
			(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROTL64(q[1 + 13], 19) ^ ROTL64(q[1 + 13], 53)) +
			(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROTL64(q[1 + 14], 28) ^ ROTL64(q[1 + 14], 59)) +
			(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROTL64(q[1 + 15], 4) ^ ROTL64(q[1 + 15], 37)) +
			((precalcf[1] + ROTL64(msg[1], 1 + 1) +
			ROTL64(msg[1 + 3], 1 + 4)) ^ hash[1 + 7]);

		q[2 + 16] = CONST_EXP2(2) +
			((precalcf[2] + ROTL64(msg[2], 2 + 1) +
			ROTL64(msg[2 + 3], 2 + 4)) ^ hash[2 + 7]);
		q[3 + 16] = CONST_EXP2(3) +
			((precalcf[3] + ROTL64(msg[3], 3 + 1) +
			ROTL64(msg[3 + 3], 3 + 4)) ^ hash[3 + 7]);
		q[4 + 16] = CONST_EXP2(4) +
			((precalcf[4] + ROTL64(msg[4], 4 + 1) +
			ROL8(msg[4 + 3])) ^ hash[4 + 7]);
		q[5 + 16] = CONST_EXP2(5) +
			((precalcf[5] + ROTL64(msg[5], 5 + 1) +
			ROTL64(msg[5 + 3], 5 + 4) - ROL16(msg[5 + 10])) ^ hash[5 + 7]);


		//#pragma unroll 3
		for (int i = 6; i < 9; i++) {
			q[i + 16] = CONST_EXP2(i) +
				((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) -
				ROTL64(msg[i - 6], (i - 6) + 1)) ^ hash[i + 7]);
		}

		q[25] = CONST_EXP2(9) +
			((vectorize((25)*(0x0555555555555555ull)) - ROTL64(msg[3], 4)) ^ hash[0]);
		q[26] = CONST_EXP2(10) +
			((vectorize((26)*(0x0555555555555555ull)) - ROTL64(msg[4], 5)) ^ hash[1]);
		q[27] = CONST_EXP2(11) +
			((vectorize((27)*(0x0555555555555555ull)) - ROTL64(msg[5], 6)) ^ hash[2]);
		q[28] = CONST_EXP2(12) +
			((vectorize((28)*(0x0555555555555555ull)) + ROL16(msg[15]) - ROTL64(msg[6], 7)) ^ hash[3]);

		q[13 + 16] = CONST_EXP2(13) +
			((precalcf[6] +
			ROTL64(msg[13 - 13], (13 - 13) + 1) - ROL8(msg[13 - 6])) ^ hash[13 - 9]);
		q[14 + 16] = CONST_EXP2(14) +
			((precalcf[7] +
			ROTL64(msg[14 - 13], (14 - 13) + 1) - ROTL64(msg[14 - 6], (14 - 6) + 1)) ^ hash[14 - 9]);
		q[15 + 16] = CONST_EXP2(15) +
			((precalcf[8] + ROL16(msg[15]) +
			ROTL64(msg[15 - 13], (15 - 13) + 1)) ^ hash[15 - 9]);

		uint2 XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
		uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

		uint2 test = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ msg[0]) + (XL64    ^ q[24] ^ q[0]);

		h[0] = test;
		h[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ msg[1]) + (XL64    ^ q[25] ^ q[1]);
		h[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ msg[2]) + (XL64    ^ q[26] ^ q[2]);
		h[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ msg[3]) + (XL64    ^ q[27] ^ q[3]);
		h[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + (XL64    ^ q[28] ^ q[4]);
		h[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ msg[5]) + (XL64    ^ q[29] ^ q[5]);
		h[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ msg[6]) + (XL64    ^ q[30] ^ q[6]);
		h[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ msg[7]) + (XL64    ^ q[31] ^ q[7]);

		h[8] = ROTL64(h[4], 9) + (XH64     ^     q[24] ^ msg[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);
		h[9] = ROTL64(h[5], 10) + (XH64     ^     q[25]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
		h[10] = ROTL64(h[6], 11) + (XH64     ^     q[26]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
		h[11] = ROTL64(h[7], 12) + (XH64     ^     q[27]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
		h[12] = ROTL64(h[0], 13) + (XH64     ^     q[28]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
		h[13] = ROTL64(h[1], 14) + (XH64     ^     q[29]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
		h[14] = ROTL64(h[2], 15) + (XH64     ^     q[30]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
		h[15] = ROL16(h[3]) + (XH64     ^     q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);

		const uint2 cmsg[16] =
		{
			0xaaaaaaa0, 0xaaaaaaaa,
			0xaaaaaaa1, 0xaaaaaaaa,
			0xaaaaaaa2, 0xaaaaaaaa,
			0xaaaaaaa3, 0xaaaaaaaa,
			0xaaaaaaa4, 0xaaaaaaaa,
			0xaaaaaaa5, 0xaaaaaaaa,
			0xaaaaaaa6, 0xaaaaaaaa,
			0xaaaaaaa7, 0xaaaaaaaa,
			0xaaaaaaa8, 0xaaaaaaaa,
			0xaaaaaaa9, 0xaaaaaaaa,
			0xaaaaaaaa, 0xaaaaaaaa,
			0xaaaaaaab, 0xaaaaaaaa,
			0xaaaaaaac, 0xaaaaaaaa,
			0xaaaaaaad, 0xaaaaaaaa,
			0xaaaaaaae, 0xaaaaaaaa,
			0xaaaaaaaf, 0xaaaaaaaa
		};

		// Final
#pragma unroll 16
		for (int i = 0; i < 16; i++)
		{
			msg[i] = cmsg[i]^h[i];
		}



		const uint2 precalc[16] =
		{
			{ 0x55555550, 0x55555555 },
			{ 0xAAAAAAA5, 0x5AAAAAAA },
			{ 0xFFFFFFFA, 0x5FFFFFFF },
			{ 0x5555554F, 0x65555555 },
			{ 0xAAAAAAA4, 0x6AAAAAAA },
			{ 0xFFFFFFF9, 0x6FFFFFFF },
			{ 0x5555554E, 0x75555555 },
			{ 0xAAAAAAA3, 0x7AAAAAAA },
			{ 0xFFFFFFF8, 0x7FFFFFFF },
			{ 0x5555554D, 0x85555555 },
			{ 0xAAAAAAA2, 0x8AAAAAAA },
			{ 0xFFFFFFF7, 0x8FFFFFFF },
			{ 0x5555554C, 0x95555555 },
			{ 0xAAAAAAA1, 0x9AAAAAAA },
			{ 0xFFFFFFF6, 0x9FFFFFFF },
			{ 0x5555554B, 0xA5555555 },
		};

		tmp = (msg[5]) - (msg[7]) + (msg[10]) + (msg[13]) + (msg[14]);
		q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[1];
		tmp = (msg[6]) - (msg[8]) + (msg[11]) + (msg[14]) - (msg[15]);
		q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[2];
		tmp = (msg[0]) + (msg[7]) + (msg[9]) - (msg[12]) + (msg[15]);
		q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[3];
		tmp = (msg[0]) - (msg[1]) + (msg[8]) - (msg[10]) + (msg[13]);
		q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[4];
		tmp = (msg[1]) + (msg[2]) + (msg[9]) - (msg[11]) - (msg[14]);
		q[4] = (SHR(tmp, 1) ^ tmp) + cmsg[5];
		tmp = (msg[3]) - (msg[2]) + (msg[10]) - (msg[12]) + (msg[15]);
		q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[6];
		tmp = (msg[4]) - (msg[0]) - (msg[3]) - (msg[11]) + (msg[13]);
		q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[7];
		tmp = (msg[1]) - (msg[4]) - (msg[5]) - (msg[12]) - (msg[14]);
		q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[8];
		tmp = (msg[2]) - (msg[5]) - (msg[6]) + (msg[13]) - (msg[15]);
		q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[9];
		tmp = (msg[0]) - (msg[3]) + (msg[6]) - (msg[7]) + (msg[14]);
		q[9] = (SHR(tmp, 1) ^ tmp) + cmsg[10];
		tmp = (msg[8]) - (msg[1]) - (msg[4]) - (msg[7]) + (msg[15]);
		q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[11];
		tmp = (msg[8]) - (msg[0]) - (msg[2]) - (msg[5]) + (msg[9]);
		q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + cmsg[12];
		tmp = (msg[1]) + (msg[3]) - (msg[6]) - (msg[9]) + (msg[10]);
		q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + cmsg[13];
		tmp = (msg[2]) + (msg[4]) + (msg[7]) + (msg[10]) + (msg[11]);
		q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + cmsg[14];
		tmp = (msg[3]) - (msg[5]) + (msg[8]) - (msg[11]) - (msg[12]);
		q[14] = (SHR(tmp, 1) ^ tmp) + cmsg[15];
		tmp = (msg[12]) - (msg[4]) - (msg[6]) - (msg[9]) + (msg[13]);
		q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + cmsg[0];

		q[0 + 16] =
			(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROTL64(q[0], 13) ^ ROTL64(q[0], 43)) +
			(SHR(q[0 + 1], 2) ^ SHL(q[0 + 1], 1) ^ ROTL64(q[0 + 1], 19) ^ ROTL64(q[0 + 1], 53)) +
			(SHR(q[0 + 2], 2) ^ SHL(q[0 + 2], 2) ^ ROTL64(q[0 + 2], 28) ^ ROTL64(q[0 + 2], 59)) +
			(SHR(q[0 + 3], 1) ^ SHL(q[0 + 3], 3) ^ ROTL64(q[0 + 3], 4) ^ ROTL64(q[0 + 3], 37)) +
			(SHR(q[0 + 4], 1) ^ SHL(q[0 + 4], 2) ^ ROTL64(q[0 + 4], 13) ^ ROTL64(q[0 + 4], 43)) +
			(SHR(q[0 + 5], 2) ^ SHL(q[0 + 5], 1) ^ ROTL64(q[0 + 5], 19) ^ ROTL64(q[0 + 5], 53)) +
			(SHR(q[0 + 6], 2) ^ SHL(q[0 + 6], 2) ^ ROTL64(q[0 + 6], 28) ^ ROTL64(q[0 + 6], 59)) +
			(SHR(q[0 + 7], 1) ^ SHL(q[0 + 7], 3) ^ ROTL64(q[0 + 7], 4) ^ ROTL64(q[0 + 7], 37)) +
			(SHR(q[0 + 8], 1) ^ SHL(q[0 + 8], 2) ^ ROTL64(q[0 + 8], 13) ^ ROTL64(q[0 + 8], 43)) +
			(SHR(q[0 + 9], 2) ^ SHL(q[0 + 9], 1) ^ ROTL64(q[0 + 9], 19) ^ ROTL64(q[0 + 9], 53)) +
			(SHR(q[0 + 10], 2) ^ SHL(q[0 + 10], 2) ^ ROTL64(q[0 + 10], 28) ^ ROTL64(q[0 + 10], 59)) +
			(SHR(q[0 + 11], 1) ^ SHL(q[0 + 11], 3) ^ ROTL64(q[0 + 11], 4) ^ ROTL64(q[0 + 11], 37)) +
			(SHR(q[0 + 12], 1) ^ SHL(q[0 + 12], 2) ^ ROTL64(q[0 + 12], 13) ^ ROTL64(q[0 + 12], 43)) +
			(SHR(q[0 + 13], 2) ^ SHL(q[0 + 13], 1) ^ ROTL64(q[0 + 13], 19) ^ ROTL64(q[0 + 13], 53)) +
			(SHR(q[0 + 14], 2) ^ SHL(q[0 + 14], 2) ^ ROTL64(q[0 + 14], 28) ^ ROTL64(q[0 + 14], 59)) +
			(SHR(q[0 + 15], 1) ^ SHL(q[0 + 15], 3) ^ ROTL64(q[0 + 15], 4) ^ ROTL64(q[0 + 15], 37)) +
			((precalc[0] + ROTL64(h[0], 0 + 1) +
			ROTL64(h[0 + 3], 0 + 4) - ROTL64(h[0 + 10], 0 + 11)) ^ cmsg[0 + 7]);
		q[1 + 16] =
			(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROTL64(q[1], 13) ^ ROTL64(q[1], 43)) +
			(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROTL64(q[1 + 1], 19) ^ ROTL64(q[1 + 1], 53)) +
			(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROTL64(q[1 + 2], 28) ^ ROTL64(q[1 + 2], 59)) +
			(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROTL64(q[1 + 3], 4) ^ ROTL64(q[1 + 3], 37)) +
			(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROTL64(q[1 + 4], 13) ^ ROTL64(q[1 + 4], 43)) +
			(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROTL64(q[1 + 5], 19) ^ ROTL64(q[1 + 5], 53)) +
			(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROTL64(q[1 + 6], 28) ^ ROTL64(q[1 + 6], 59)) +
			(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROTL64(q[1 + 7], 4) ^ ROTL64(q[1 + 7], 37)) +
			(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROTL64(q[1 + 8], 13) ^ ROTL64(q[1 + 8], 43)) +
			(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROTL64(q[1 + 9], 19) ^ ROTL64(q[1 + 9], 53)) +
			(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROTL64(q[1 + 10], 28) ^ ROTL64(q[1 + 10], 59)) +
			(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROTL64(q[1 + 11], 4) ^ ROTL64(q[1 + 11], 37)) +
			(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROTL64(q[1 + 12], 13) ^ ROTL64(q[1 + 12], 43)) +
			(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROTL64(q[1 + 13], 19) ^ ROTL64(q[1 + 13], 53)) +
			(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROTL64(q[1 + 14], 28) ^ ROTL64(q[1 + 14], 59)) +
			(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROTL64(q[1 + 15], 4) ^ ROTL64(q[1 + 15], 37)) +
			((precalc[1] + ROTL64(h[1], 1 + 1) +
			ROTL64(h[1 + 3], 1 + 4) - ROTL64(h[1 + 10], 1 + 11)) ^ cmsg[1 + 7]);

		q[2 + 16] = CONST_EXP2(2) +
			((precalc[2] + ROTL64(h[2], 2 + 1) +
			ROTL64(h[2 + 3], 2 + 4) - ROTL64(h[2 + 10], 2 + 11)) ^ cmsg[2 + 7]);
		q[3 + 16] = CONST_EXP2(3) +
			((precalc[3] + ROTL64(h[3], 3 + 1) +
			ROTL64(h[3 + 3], 3 + 4) - ROTL64(h[3 + 10], 3 + 11)) ^ cmsg[3 + 7]);
		q[4 + 16] = CONST_EXP2(4) +
			((precalc[4] + ROTL64(h[4], 4 + 1) +
			ROL8(h[4 + 3]) - ROTL64(h[4 + 10], 4 + 11)) ^ cmsg[4 + 7]);
		q[5 + 16] = CONST_EXP2(5) +
			((precalc[5] + ROTL64(h[5], 5 + 1) +
			ROTL64(h[5 + 3], 5 + 4) - ROL16(h[5 + 10])) ^ cmsg[5 + 7]);


		q[6 + 16] = CONST_EXP2(6) +
			((precalc[6] + ROTL64(h[6], 6 + 1) +
			ROTL64(h[6 + 3], 6 + 4) - ROTL64(h[6 - 6], (6 - 6) + 1)) ^ cmsg[6 + 7]);
		q[7 + 16] = CONST_EXP2(7) +
			((precalc[7] + ROL8(h[7]) +
			ROTL64(h[7 + 3], 7 + 4) - ROTL64(h[7 - 6], (7 - 6) + 1)) ^ cmsg[7 + 7]);
		q[8 + 16] = CONST_EXP2(8) +
			((precalc[8] + ROTL64(h[8], 8 + 1) +
			ROTL64(h[8 + 3], 8 + 4) - ROTL64(h[8 - 6], (8 - 6) + 1)) ^ cmsg[8 + 7]);

		q[9 + 16] = CONST_EXP2(9) +
			((precalc[9] + ROTL64(h[9], 9 + 1) +
			ROTL64(h[9 + 3], 9 + 4) - ROTL64(h[9 - 6], (9 - 6) + 1)) ^ cmsg[9 - 9]);
		q[10 + 16] = CONST_EXP2(10) +
			((precalc[10] + ROTL64(h[10], 10 + 1) +
			ROTL64(h[10 + 3], 10 + 4) - ROTL64(h[10 - 6], (10 - 6) + 1)) ^ cmsg[10 - 9]);
		q[11 + 16] = CONST_EXP2(11) +
			((precalc[11] + ROTL64(h[11], 11 + 1) +
			ROTL64(h[11 + 3], 11 + 4) - ROTL64(h[11 - 6], (11 - 6) + 1)) ^ cmsg[11 - 9]);
		q[12 + 16] = CONST_EXP2(12) +
			((precalc[12] + ROTL64(h[12], 12 + 1) +
			ROL16(h[12 + 3]) - ROTL64(h[12 - 6], (12 - 6) + 1)) ^ cmsg[12 - 9]);



		q[13 + 16] = CONST_EXP2(13) +
			((precalc[13] + ROTL64(h[13], 13 + 1) +
			ROTL64(h[13 - 13], (13 - 13) + 1) - ROL8(h[13 - 6])) ^ cmsg[13 - 9]);
		q[14 + 16] = CONST_EXP2(14) +
			((precalc[14] + ROTL64(h[14], 14 + 1) +
			ROTL64(h[14 - 13], (14 - 13) + 1) - ROTL64(h[14 - 6], (14 - 6) + 1)) ^ cmsg[14 - 9]);
		q[15 + 16] = CONST_EXP2(15) +
			((precalc[15] + ROL16(h[15]) +
			ROTL64(h[15 - 13], (15 - 13) + 1) - ROTL64(h[15 - 6], (15 - 6) + 1)) ^ cmsg[15 - 9]);

		XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
		XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

		msg[4] = (SHR(XH64, 3) ^ q[20] ^ h[4]) + (XL64    ^ q[28] ^ q[4]);
		msg[8] = ROTL64(msg[4], 9) + (XH64     ^     q[24] ^ h[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);

		inpHash[0] = devectorize(msg[0 + 8]);

		if (((msg[8].x) & 0x8)) return;

		msg[0] = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ h[0]) + (XL64    ^ q[24] ^ q[0]);
		msg[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ h[1]) + (XL64    ^ q[25] ^ q[1]);
		msg[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ h[2]) + (XL64    ^ q[26] ^ q[2]);
		msg[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ h[3]) + (XL64    ^ q[27] ^ q[3]);
		msg[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ h[5]) + (XL64    ^ q[29] ^ q[5]);
		msg[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ h[6]) + (XL64    ^ q[30] ^ q[6]);
		msg[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ h[7]) + (XL64    ^ q[31] ^ q[7]);

		msg[9] = ROTL64(msg[5], 10) + (XH64     ^     q[25] ^ h[9]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
		msg[10] = ROTL64(msg[6], 11) + (XH64     ^     q[26] ^ h[10]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
		msg[11] = ROTL64(msg[7], 12) + (XH64     ^     q[27] ^ h[11]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
		msg[12] = ROTL64(msg[0], 13) + (XH64     ^     q[28] ^ h[12]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
		msg[13] = ROTL64(msg[1], 14) + (XH64     ^     q[29] ^ h[13]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
		msg[14] = ROTL64(msg[2], 15) + (XH64     ^     q[30] ^ h[14]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
		msg[15] = ROL16(msg[3]) + (XH64     ^     q[31] ^ h[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);

		inpHash[1] = devectorize(msg[1 + 8]);
		inpHash[2] = devectorize(msg[2 + 8]);
		inpHash[3] = devectorize(msg[3 + 8]);
		inpHash[4] = devectorize(msg[4 + 8]);
		inpHash[5] = devectorize(msg[5 + 8]);
		inpHash[6] = devectorize(msg[6 + 8]);
		inpHash[7] = devectorize(msg[7 + 8]);
	}
}


__global__ __launch_bounds__(256, 2)
void quark_bmw512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
    const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        const uint32_t nounce = startNounce + thread;

        // Init
		uint2 __align__(64) h[16] = {
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};

		uint2 message[16];
#pragma unroll 16
        for(int i=0;i<16;i++)
			message[i] = vectorize(c_PaddedMessage80[i]);

		message[9].y = cuda_swab32(nounce);	//REPLACE_HIWORD(message[9], cuda_swab32(nounce));
        Compression512(message, h);

#pragma unroll 16
        for(int i=0;i<16;i++)
			message[i] = make_uint2(0xaaaaaaa0+i,0xaaaaaaaa);


		Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * thread];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = devectorize(message[i+8]);
    }
}

// Bmw512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_bmw512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	uint64_t *message = (uint64_t*)PaddedMessage;
	// Padding einf�gen (Byteorder?!?)
	message[10] = SPH_C64(0x80);
	// L�nge (in Bits, d.h. 80 Byte * 8 = 640 Bits
	message[15] = SPH_C64(640);

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void quark_bmw512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 32;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_bmw512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}
__host__ void quark_bmw512_cpu_hash_64_quark(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 32;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	quark_bmw512_gpu_hash_64_quark << <grid, block >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}



__host__ void quark_bmw512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{
    const uint32_t threadsperblock = 128;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_bmw512_gpu_hash_80<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
}

