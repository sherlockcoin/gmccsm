#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#ifdef _MSC_VER
#define UINT2(x,y) { x, y }
#else
#define UINT2(x,y) (uint2) { x, y }
#endif

__constant__ uint2 c_keccak_round_constants35[24] = {
		{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
		{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
		{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
		{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
		{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
		{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
		{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
		{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
		{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
		{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};
#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))


static __device__ __forceinline__ void
keccak_block_35(uint2 *s)
{
	uint8_t i;

	#pragma unroll 3
	for (i = 0; i < 24; ++i)
	{
		uint2 bc[5], tmpxor[5], tmp1, tmp2;

		#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL2(s[19] ^ bc[3], 8);
		s[19] = ROL2(s[23] ^ bc[2], 56);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= c_keccak_round_constants35[i];
	}
}

static __device__ __forceinline__ void keccak_block_35_final(uint2 *s)
{
	int i = 0;

	#pragma unroll 3
	for (i = 0; i < 23; i++)
	{
		uint2 bc[5], tmpxor[5], tmp1, tmp2;

#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL2(s[19] ^ bc[3], 8);
		s[19] = ROL2(s[23] ^ bc[2], 56);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= c_keccak_round_constants35[i];
	}
	uint2 t[5];
	t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
	t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
	t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
	t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
	t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

	s[0] ^= t[4] ^ ROL2(t[1], 1);
	s[18] ^= t[2] ^ ROL2(t[4], 1);
	s[24] ^= t[3] ^ ROL2(t[0], 1);

	s[3] = ROL2(s[18], 21) ^ ((~ROL2(s[24], 14)) & s[0]);
}

__global__  __launch_bounds__(128, 4)
void quark_keccak512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint64_t *inpHash = &g_hash[8 * hashPosition];

        uint2 keccak_gpu_state[25];
#pragma unroll
		for (int i = 0; i<8; i++)
		{
			keccak_gpu_state[i] = vectorize(inpHash[i]);
		}
		keccak_gpu_state[8] = make_uint2(0x00000001UL, 0x80000000);	//vectorize(0x8000000000000001ULL);

#pragma unroll
        for (int i=9; i<25; i++)
		{
			keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_block_35(keccak_gpu_state);

#pragma unroll
        for(int i=0;i<8;i++)
			inpHash[i] = devectorize(keccak_gpu_state[i]);
    }
}

__global__  __launch_bounds__(128, 4)
void quark_keccak512_gpu_hash_64_final(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		uint2 keccak_gpu_state[25];
#pragma unroll
		for (int i = 0; i<8; i++)
		{
			keccak_gpu_state[i] = vectorize(inpHash[i]);
		}
		keccak_gpu_state[8] = make_uint2(0x00000001UL, 0x80000000);	//vectorize(0x8000000000000001ULL);

#pragma unroll
		for (int i = 9; i<25; i++)
		{
			keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_block_35_final(keccak_gpu_state);

		inpHash[3] = devectorize(keccak_gpu_state[3]);
	}
}

__host__ void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
    const uint32_t threadsperblock = 32;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_keccak512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}

__host__ void quark_keccak512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 32;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	quark_keccak512_gpu_hash_64_final << <grid, block >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}
